#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <emmintrin.h>
#include <sys/time.h>
#include "GPURiemannPlugin.h"


void GPURiemannPlugin::input(std::string file) {
   std::ifstream infile(file.c_str(), std::ios::in);
   infile >> N;
}

void GPURiemannPlugin::run() {
	  //size of the arrays in bytes
  size_t size = N * sizeof(double);

  //allocate array on host and device
  a_h = (double *)malloc(size);
  hipMalloc((void **) &a_d, size);

  //do calculation on device
  int block_size = 1024;
  int n_blocks = N/block_size + (N % block_size == 0 ? 0:1);
  integratorKernel <<< n_blocks, block_size >>> (a_d, N);

  //copy results from device to host
  hipMemcpy(a_h, a_d, sizeof(double)*N, hipMemcpyDeviceToHost);

  //add results
  sum = 0;
  for (int i = 0; i < N; i++) sum += a_h[i];
  sum *= 1 / (double)N;;

}


void GPURiemannPlugin::output(std::string file) {
  //clean up
	printf("%.54lf\n", sum);
  free(a_h);
  hipFree(a_d);
}


PluginProxy<GPURiemannPlugin> GPURiemannPluginProxy = PluginProxy<GPURiemannPlugin>("GPURiemann", PluginManager::getInstance());

